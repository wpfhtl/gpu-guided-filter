#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>

#include "box_filter.h"
#include "math_kernels.h"

__device__ void compute_cov_var(float4 *mean_Ip, float4 *mean_II, float4 *mean_I,
        float4 *mean_p, float4 *var_I, float4 *cov_Ip, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = y * width + x; 
    float4 m_I = mean_I[idx];
    var_I[idx] = mean_II[idx] - m_I * m_I;
    cov_Ip[idx] = mean_Ip[idx] - m_I * mean_p[idx];
}

__device__ void compute_ab(float4 *var_I, float4 *cov_Ip, float4 *mean_I,
        float4 *mean_p, float4 *a, float4 *b, float eps, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = y * width + x; 
    float4 a_ = cov_Ip[idx] / (var_I[idx] + eps);
    a[idx] = a_;
    b[idx] = mean_p[idx] - a_ * mean_I[idx];
}

__device__ void compute_q(float4 *in, float4 *mean_a, float4 *mean_b, float4 *q,
        int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = y * width + x; 
    float4 im_ = in[idx];
    q[idx] = mean_a[idx] * im_ + mean_b[idx];
}

__global__ void mean_kernel(float4* d_input,
        float4 *d_p,
        float4 *mean_I,
        float4 *mean_p,
        float4 *mean_Ip,
        float4 *mean_II,
        float4 *d_tmp,
        float4 *d_tmp2,
        int width, int height,
        float eps)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        box_filter(d_input, mean_I, width, height);
        box_filter(d_p, mean_p, width, height);
        box_filter(d_tmp, mean_Ip, width, height);
        box_filter(d_tmp2, mean_II, width, height);
    }
}

__global__ void cov_var_ab_kernel(float4* d_input,
        float4 *mean_I,
        float4 *mean_p,
        float4 *mean_Ip,
        float4 *mean_II,
        float4 *var_I,
        float4 *cov_Ip,
        float4 *a, 
        float4 *b,
        float4 *mean_a,
        float4 *mean_b,
        int width, int height,
        float eps)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        compute_cov_var(mean_Ip, mean_II, mean_I, mean_p, var_I, cov_Ip, width, height);
        compute_ab(var_I, cov_Ip, mean_I, mean_p, a, b, eps, width, height);
    }
}

__global__ void output_kernel(float4* d_input,
        float4 *d_p,
        float4 *d_q,
        float4 *a, 
        float4 *b,
        float4 *mean_a,
        float4 *mean_b,
        int width, int height,
        float eps)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        box_filter(a, mean_a, width, height);
        box_filter(b, mean_b, width, height);
        compute_q(d_p, mean_a, mean_b, d_q, width, height);
    }
}

#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void guided_filter_cuda(float4 *h_input,
        float4 *h_p,
        float4 *h_output,
        float4 *h_tmp,
        float4 *h_tmp2,
        int width, int height,
        float eps)
{

    const int n = width * height * sizeof(float4);

    float4 *d_input, *d_p, *d_output, *d_mean_I, *d_mean_p, *d_mean_Ip,
          *d_mean_II, *d_var_I, *d_cov_Ip, *d_a, *d_b, *d_mean_a,
          *d_mean_b, *d_tmp, *d_tmp2;

    checkCudaErrors(hipMalloc<float4>(&d_input, n));
    checkCudaErrors(hipMalloc<float4>(&d_p, n));
    checkCudaErrors(hipMalloc<float4>(&d_output, n));
    checkCudaErrors(hipMalloc<float4>(&d_mean_I, n));
    checkCudaErrors(hipMalloc<float4>(&d_mean_p, n));
    checkCudaErrors(hipMalloc<float4>(&d_mean_Ip, n));
    checkCudaErrors(hipMalloc<float4>(&d_mean_II, n));
    checkCudaErrors(hipMalloc<float4>(&d_var_I, n));
    checkCudaErrors(hipMalloc<float4>(&d_cov_Ip, n));
    checkCudaErrors(hipMalloc<float4>(&d_a, n));
    checkCudaErrors(hipMalloc<float4>(&d_b, n));
    checkCudaErrors(hipMalloc<float4>(&d_mean_a, n));
    checkCudaErrors(hipMalloc<float4>(&d_mean_b, n));
    checkCudaErrors(hipMalloc<float4>(&d_tmp, n));
    checkCudaErrors(hipMalloc<float4>(&d_tmp2, n));

    checkCudaErrors(hipMemcpy(d_input, h_input, n, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_p, h_p, n, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_output, h_output, n, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_tmp, h_tmp, n, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_tmp2, h_tmp2, n, hipMemcpyHostToDevice));

    int GRID_W = ceil(width / (float)TILE_W);
    int GRID_H = ceil(height / (float)TILE_H);

    const dim3 block(TILE_W, TILE_H);
    const dim3 grid(GRID_W, GRID_H);

    printf("grid_w: %d\n", grid.x);
    printf("grid_h: %d\n", grid.y);
    printf("block_w: %d\n", block.x);
    printf("block_h: %d\n", block.y);

    mean_kernel<<<grid, block>>>(d_input, d_p, d_mean_I, d_mean_p, d_mean_Ip,
            d_mean_II, d_tmp, d_tmp2, width, height, eps);

    hipDeviceSynchronize();

    cov_var_ab_kernel<<<grid, block>>>(d_input, d_mean_I, d_mean_p, d_mean_Ip,
        d_mean_II, d_var_I, d_cov_Ip, d_a, d_b, d_mean_a, d_mean_b, 
        width, height, eps);

    hipDeviceSynchronize();

    output_kernel<<<grid, block>>>(d_input, d_p, d_output, d_a, d_b,
            d_mean_a, d_mean_b, width, height, eps);

    hipDeviceSynchronize();

    auto error = hipGetLastError();
    if (error != hipSuccess)
        printf("An error occured with CUDA: %s\n", hipGetErrorString(error));

    checkCudaErrors(hipMemcpy(h_output, d_output, n, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_input));
    checkCudaErrors(hipFree(d_p));
    checkCudaErrors(hipFree(d_output));
    checkCudaErrors(hipFree(d_mean_I));
    checkCudaErrors(hipFree(d_mean_p));
    checkCudaErrors(hipFree(d_mean_Ip));
    checkCudaErrors(hipFree(d_mean_II));
    checkCudaErrors(hipFree(d_var_I));
    checkCudaErrors(hipFree(d_cov_Ip));
    checkCudaErrors(hipFree(d_a));
    checkCudaErrors(hipFree(d_b));
    checkCudaErrors(hipFree(d_mean_a));
    checkCudaErrors(hipFree(d_mean_b));
    checkCudaErrors(hipFree(d_tmp));
    checkCudaErrors(hipFree(d_tmp2));
}
